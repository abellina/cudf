#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <cstddef>
#include <numeric>

namespace cudf {
namespace {

// align all column size allocations to this boundary so that all output column buffers
// start at that alignment.
static constexpr std::size_t split_align = 64;

/**
 * @brief Struct which contains information on a source buffer.
 *
 * The definition of "buffer" used throughout this module is a component piece of a
 * cudf column. So for example, a fixed-width column with validity would have 2 associated
 * buffers : the data itself and the validity buffer.  contiguous_split operates by breaking
 * each column up into it's individual components and copying each one as a separate kernel
 * block.
 */
struct src_buf_info {
  src_buf_info(cudf::type_id _type,
               const int* _offsets,
               int _offset_stack_pos,
               int _parent_offsets_index,
               bool _is_validity,
               size_type _column_offset)
    : type(_type),
      offsets(_offsets),
      offset_stack_pos(_offset_stack_pos),
      parent_offsets_index(_parent_offsets_index),
      is_validity(_is_validity),
      column_offset(_column_offset)
  {
  }

  cudf::type_id type;
  const int* offsets;        // a pointer to device memory offsets if I am an offset buffer
  int offset_stack_pos;      // position in the offset stack buffer
  int parent_offsets_index;  // immediate parent that has offsets, or -1 if none
  bool is_validity;          // if I am a validity buffer
  size_type column_offset;   // offset in the case of a sliced column
};

/**
 * @brief Struct which contains information on a destination buffer.
 *
 * Similar to src_buf_info, dst_buf_info contains information on a destination buffer we
 * are going to copy to.  If we have N input buffers (which come from X columns), and
 * M partitions, then we have N*M destination buffers.
 */
struct dst_buf_info {
  // constant across all copy commands for this buffer
  std::size_t buf_size;  // total size of buffer, including padding
  int num_elements;      // # of elements to be copied
  int element_size;      // size of each element in bytes
  int num_rows;  // # of rows to be copied(which may be different from num_elements in the case of
                 // validity or offset buffers)

  int src_element_index;   // element index to start reading from from my associated source buffer
  std::size_t dst_offset;  // my offset into the per-partition allocation
  int value_shift;         // amount to shift values down by (for offset buffers)
  int bit_shift;           // # of bits to shift right by (for validity buffers)
  size_type valid_count;   // validity count for this block of work

  int src_buf_index;       // source buffer index
  int dst_buf_index;       // destination buffer index
};

/**
 * @brief Copy a single buffer of column data, shifting values (for offset columns),
 * and validity (for validity buffers) as necessary.
 *
 * Copies a single partition of a source column buffer to a destination buffer. Shifts
 * element values by value_shift in the case of a buffer of offsets (value_shift will
 * only ever be > 0 in that case).  Shifts elements bitwise by bit_shift in the case of
 * a validity buffer (bif_shift will only ever be > 0 in that case).  This function assumes
 * value_shift and bit_shift will never be > 0 at the same time.
 *
 * This function expects:
 * - src may be a misaligned address
 * - dst must be an aligned address
 *
 * This function always does the ALU work related to value_shift and bit_shift because it is
 * entirely memory-bandwidth bound.
 *
 * @param dst Destination buffer
 * @param src Source buffer
 * @param t Thread index
 * @param num_elements Number of elements to copy
 * @param element_size Size of each element in bytes
 * @param src_element_index Element index to start copying at
 * @param stride Size of the kernel block
 * @param value_shift Shift incoming 4-byte offset values down by this amount
 * @param bit_shift Shift incoming data right by this many bits
 * @param num_rows Number of rows being copied
 * @param valid_count Optional pointer to a value to store count of set bits
 */
template <int block_size>
__device__ void copy_buffer(uint8_t* __restrict__ dst,
                            uint8_t const* __restrict__ src,
                            int t,
                            std::size_t num_elements,
                            std::size_t element_size,
                            std::size_t src_element_index,
                            uint32_t stride,
                            int value_shift,
                            int bit_shift,
                            std::size_t num_rows,
                            size_type* valid_count)
{
  src += (src_element_index * element_size);

  size_type thread_valid_count = 0;

  // handle misalignment. read 16 bytes in 4 byte reads. write in a single 16 byte store.
  std::size_t const num_bytes = num_elements * element_size;
  // how many bytes we're misaligned from 4-byte alignment
  uint32_t const ofs = reinterpret_cast<uintptr_t>(src) % 4;
  std::size_t pos    = t * 16;
  stride *= 16;
  while (pos + 20 <= num_bytes) {
    // read from the nearest aligned address.
    const uint32_t* in32 = reinterpret_cast<const uint32_t*>((src + pos) - ofs);
    uint4 v              = uint4{in32[0], in32[1], in32[2], in32[3]};
    if (ofs || bit_shift) {
      v.x = __funnelshift_r(v.x, v.y, ofs * 8 + bit_shift);
      v.y = __funnelshift_r(v.y, v.z, ofs * 8 + bit_shift);
      v.z = __funnelshift_r(v.z, v.w, ofs * 8 + bit_shift);
      v.w = __funnelshift_r(v.w, in32[4], ofs * 8 + bit_shift);
    }
    v.x -= value_shift;
    v.y -= value_shift;
    v.z -= value_shift;
    v.w -= value_shift;
    reinterpret_cast<uint4*>(dst)[pos / 16] = v;
    if (valid_count) {
      thread_valid_count += (__popc(v.x) + __popc(v.y) + __popc(v.z) + __popc(v.w));
    }
    pos += stride;
  }

  // copy trailing bytes
  if (t == 0) {
    std::size_t remainder;
    if (num_bytes < 16) {
      remainder = num_bytes;
    } else {
      std::size_t const last_bracket = (num_bytes / 16) * 16;
      remainder                      = num_bytes - last_bracket;
      if (remainder < 4) {
        // we had less than 20 bytes for the last possible 16 byte copy, so copy 16 + the extra
        remainder += 16;
      }
    }

    // if we're performing a value shift (offsets), or a bit shift (validity) the # of bytes and
    // alignment must be a multiple of 4. value shifting and bit shifting are mutually exclusive
    // and will never both be true at the same time.
    if (value_shift || bit_shift) {
      std::size_t idx = (num_bytes - remainder) / 4;
      uint32_t v = remainder > 0 ? (reinterpret_cast<uint32_t const*>(src)[idx] - value_shift) : 0;

      constexpr size_type rows_per_element = 32;
      auto const have_trailing_bits = ((num_elements * rows_per_element) - num_rows) < bit_shift;
      while (remainder) {
        // if we're at the very last word of a validity copy, we do not always need to read the next
        // word to get the final trailing bits.
        auto const read_trailing_bits = bit_shift > 0 && remainder == 4 && have_trailing_bits;
        uint32_t const next           = (read_trailing_bits || remainder > 4)
                                          ? (reinterpret_cast<uint32_t const*>(src)[idx + 1] - value_shift)
                                          : 0;

        uint32_t const val = (v >> bit_shift) | (next << (32 - bit_shift));
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint32_t*>(dst)[idx] = val;
        v                                     = next;
        idx++;
        remainder -= 4;
      }
    } else {
      while (remainder) {
        std::size_t const idx = num_bytes - remainder--;
        uint32_t const val    = reinterpret_cast<uint8_t const*>(src)[idx];
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint8_t*>(dst)[idx] = val;
      }
    }
  }

  if (valid_count) {
    if (num_bytes == 0) {
      if (!t) { *valid_count = 0; }
    } else {
      using BlockReduce = hipcub::BlockReduce<size_type, block_size>;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      size_type block_valid_count{BlockReduce(temp_storage).Sum(thread_valid_count)};
      if (!t) {
        // we may have copied more bits than there are actual rows in the output.
        // so we need to subtract off the count of any bits that shouldn't have been
        // considered during the copy step.
        std::size_t const max_row    = (num_bytes * 8);
        std::size_t const slack_bits = max_row > num_rows ? max_row - num_rows : 0;
        auto const slack_mask        = set_most_significant_bits(slack_bits);
        if (slack_mask > 0) {
          uint32_t const last_word = reinterpret_cast<uint32_t*>(dst + (num_bytes - 4))[0];
          block_valid_count -= __popc(last_word & slack_mask);
        }
        *valid_count = block_valid_count;
      }
    }
  }
}

/**
 * @brief Kernel which copies data from multiple source buffers to multiple
 * destination buffers.
 *
 * When doing a contiguous_split on X columns comprising N total internal buffers
 * with M splits, we end up having to copy N*M source/destination buffer pairs.
 * These logical copies are further subdivided to distribute the amount of work
 * to be done as evenly as possible across the multiprocessors on the device.
 * This kernel is arranged such that each block copies 1 source/destination pair.
 *
 * @param src_bufs Input source buffers
 * @param dst_bufs Destination buffers
 * @param buf_info Information on the range of values to be copied for each destination buffer.
 */
template <int block_size>
__global__ void copy_partitions(uint8_t const** src_bufs,
                                uint8_t** dst_bufs,
                                dst_buf_info* buf_info)
{
  auto const buf_index     = blockIdx.x;
  auto const src_buf_index = buf_info[buf_index].src_buf_index;
  auto const dst_buf_index = buf_info[buf_index].dst_buf_index;

  // copy, shifting offsets and validity bits as needed
  copy_buffer<block_size>(
    dst_bufs[dst_buf_index] + buf_info[buf_index].dst_offset,
    src_bufs[src_buf_index],
    threadIdx.x,
    buf_info[buf_index].num_elements,
    buf_info[buf_index].element_size,
    buf_info[buf_index].src_element_index,
    blockDim.x,
    buf_info[buf_index].value_shift,
    buf_info[buf_index].bit_shift,
    buf_info[buf_index].num_rows,
    buf_info[buf_index].valid_count > 0 ? &buf_info[buf_index].valid_count : nullptr);
}

// The block of functions below are all related:
//
// compute_offset_stack_size()
// setup_src_buf_data()
// count_src_bufs()
// setup_source_buf_info()
// build_output_columns()
//
// Critically, they all traverse the hierarchy of source columns and their children
// in a specific order to guarantee they produce various outputs in a consistent
// way.  For example, setup_src_buf_info() produces a series of information
// structs that must appear in the same order that setup_src_buf_data() produces
// buffers.
//
// So please be careful if you change the way in which these functions and
// functors traverse the hierarchy.

/**
 * @brief Returns whether or not the specified type is a column that contains offsets.
 */
bool is_offset_type(type_id id) { return (id == type_id::STRING or id == type_id::LIST); }

/**
 * @brief Compute total device memory stack size needed to process nested
 * offsets per-output buffer.
 *
 * When determining the range of rows to be copied for each output buffer
 * we have to recursively apply the stack of offsets from our parent columns
 * (lists or strings).  We want to do this computation on the gpu because offsets
 * are stored in device memory.  However we don't want to do recursion on the gpu, so
 * each destination buffer gets a "stack" of space to work with equal in size to
 * it's offset nesting depth.  This function computes the total size of all of those
 * stacks.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param offset_depth Current offset nesting depth
 *
 * @returns Total offset stack size needed for this range of columns.
 */
template <typename InputIter>
std::size_t compute_offset_stack_size(InputIter begin, InputIter end, int offset_depth = 0)
{
  return std::accumulate(begin, end, 0, [offset_depth](auto stack_size, column_view const& col) {
    auto const num_buffers = 1 + (col.nullable() ? 1 : 0);
    return stack_size + (offset_depth * num_buffers) +
           compute_offset_stack_size(
             col.child_begin(), col.child_end(), offset_depth + is_offset_type(col.type().id()));
  });
}

/**
 * @brief Retrieve all buffers for a range of source columns.
 *
 * Retrieve the individual buffers that make up a range of input columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param out_buf Iterator into output buffer infos
 *
 * @returns next output buffer iterator
 */
template <typename InputIter, typename OutputIter>
OutputIter setup_src_buf_data(InputIter begin, InputIter end, OutputIter out_buf)
{
  std::for_each(begin, end, [&out_buf](column_view const& col) {
    if (col.nullable()) {
      *out_buf = reinterpret_cast<uint8_t const*>(col.null_mask());
      out_buf++;
    }
    // NOTE: we're always returning the base pointer here.  column-level offset is accounted
    // for later. Also, for some column types (string, list, struct) this pointer will be null
    // because there is no associated data with the root column.
    *out_buf = col.head<uint8_t>();
    out_buf++;

    out_buf = setup_src_buf_data(col.child_begin(), col.child_end(), out_buf);
  });
  return out_buf;
}

/**
 * @brief Count the total number of source buffers we will be copying
 * from.
 *
 * This count includes buffers for all input columns. For example a
 * fixed-width column with validity would be 2 buffers (data, validity).
 * A string column with validity would be 3 buffers (chars, offsets, validity).
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 *
 * @returns total number of source buffers for this range of columns
 */
template <typename InputIter>
size_type count_src_bufs(InputIter begin, InputIter end)
{
  auto buf_iter = thrust::make_transform_iterator(begin, [](column_view const& col) {
    auto children_counts = count_src_bufs(col.child_begin(), col.child_end());
    return 1 + (col.nullable() ? 1 : 0) + children_counts;
  });
  return std::accumulate(buf_iter, buf_iter + std::distance(begin, end), 0);
}

/**
 * @brief Computes source buffer information for the copy kernel.
 *
 * For each input column to be split we need to know several pieces of information
 * in the copy kernel.  This function traverses the input columns and prepares this
 * information for the gpu.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param head Beginning of source buffer info array
 * @param current Current source buffer info to be written to
 * @param offset_stack_pos Integer representing our current offset nesting depth
 * (how many list or string levels deep we are)
 * @param parent_offset_index Index into src_buf_info output array indicating our nearest
 * containing list parent. -1 if we have no list parent
 * @param offset_depth Current offset nesting depth (how many list levels deep we are)
 *
 * @returns next src_buf_output after processing this range of input columns
 */
// setup source buf info
template <typename InputIter>
std::pair<src_buf_info*, size_type> setup_source_buf_info(InputIter begin,
                                                          InputIter end,
                                                          src_buf_info* head,
                                                          src_buf_info* current,
                                                          int offset_stack_pos    = 0,
                                                          int parent_offset_index = -1,
                                                          int offset_depth        = 0);

/**
 * @brief Functor that builds source buffer information based on input columns.
 *
 * Called by setup_source_buf_info to build information for a single source column.  This function
 * will recursively call setup_source_buf_info in the case of nested types.
 */
struct buf_info_functor {
  src_buf_info* head;

  template <typename T>
  std::pair<src_buf_info*, size_type> operator()(column_view const& col,
                                                 src_buf_info* current,
                                                 int offset_stack_pos,
                                                 int parent_offset_index,
                                                 int offset_depth)
  {
    if (col.nullable()) {
      std::tie(current, offset_stack_pos) =
        add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
    }

    // info for the data buffer
    *current = src_buf_info(
      col.type().id(), nullptr, offset_stack_pos, parent_offset_index, false, col.offset());

    return {current + 1, offset_stack_pos + offset_depth};
  }

  template <typename T, typename... Args>
  std::enable_if_t<std::is_same_v<T, cudf::dictionary32>, std::pair<src_buf_info*, size_type>>
  operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type");
  }

 private:
  std::pair<src_buf_info*, size_type> add_null_buffer(column_view const& col,
                                                      src_buf_info* current,
                                                      int offset_stack_pos,
                                                      int parent_offset_index,
                                                      int offset_depth)
  {
    // info for the validity buffer
    *current = src_buf_info(
      type_id::INT32, nullptr, offset_stack_pos, parent_offset_index, true, col.offset());

    return {current + 1, offset_stack_pos + offset_depth};
  }
};

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::string_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth)
{
  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // string columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *current = src_buf_info(
    type_id::STRING, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  current++;
  offset_stack_pos += offset_depth;

  // string columns don't necessarily have children
  if (col.num_children() > 0) {
    CUDF_EXPECTS(col.num_children() == 2, "Encountered malformed string column");
    strings_column_view scv(col);

    // info for the offsets buffer
    auto offset_col = current;
    CUDF_EXPECTS(not scv.offsets().nullable(), "Encountered nullable string offsets column");
    *current = src_buf_info(type_id::INT32,
                            // note: offsets can be null in the case where the string column
                            // has been created with empty_like().
                            scv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                            offset_stack_pos,
                            parent_offset_index,
                            false,
                            col.offset());

    current++;
    offset_stack_pos += offset_depth;

    // since we are crossing an offset boundary, calculate our new depth and parent offset index.
    offset_depth++;
    parent_offset_index = offset_col - head;

    // prevent appending buf_info for non-existent chars buffer
    CUDF_EXPECTS(not scv.chars().nullable(), "Encountered nullable string chars column");

    // info for the chars buffer
    *current = src_buf_info(
      type_id::INT8, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
    current++;
    offset_stack_pos += offset_depth;
  }

  return {current, offset_stack_pos};
}

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::list_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth)
{
  lists_column_view lcv(col);

  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // list columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *current = src_buf_info(
    type_id::LIST, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  current++;
  offset_stack_pos += offset_depth;

  CUDF_EXPECTS(col.num_children() == 2, "Encountered malformed list column");

  // info for the offsets buffer
  auto offset_col = current;
  *current        = src_buf_info(type_id::INT32,
                          // note: offsets can be null in the case where the lists column
                          // has been created with empty_like().
                          lcv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                          offset_stack_pos,
                          parent_offset_index,
                          false,
                          col.offset());
  current++;
  offset_stack_pos += offset_depth;

  // since we are crossing an offset boundary, calculate our new depth and parent offset index.
  offset_depth++;
  parent_offset_index = offset_col - head;

  return setup_source_buf_info(col.child_begin() + 1,
                               col.child_end(),
                               head,
                               current,
                               offset_stack_pos,
                               parent_offset_index,
                               offset_depth);
}

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::struct_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth)
{
  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // struct columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *current = src_buf_info(
    type_id::STRUCT, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  current++;
  offset_stack_pos += offset_depth;

  // recurse on children
  cudf::structs_column_view scv(col);
  std::vector<column_view> sliced_children;
  sliced_children.reserve(scv.num_children());
  std::transform(thrust::make_counting_iterator(0),
                 thrust::make_counting_iterator(scv.num_children()),
                 std::back_inserter(sliced_children),
                 [&scv](size_type child_index) { return scv.get_sliced_child(child_index); });
  return setup_source_buf_info(sliced_children.begin(),
                               sliced_children.end(),
                               head,
                               current,
                               offset_stack_pos,
                               parent_offset_index,
                               offset_depth);
}

template <typename InputIter>
std::pair<src_buf_info*, size_type> setup_source_buf_info(InputIter begin,
                                                          InputIter end,
                                                          src_buf_info* head,
                                                          src_buf_info* current,
                                                          int offset_stack_pos,
                                                          int parent_offset_index,
                                                          int offset_depth)
{
  std::for_each(begin, end, [&](column_view const& col) {
    std::tie(current, offset_stack_pos) = cudf::type_dispatcher(col.type(),
                                                                buf_info_functor{head},
                                                                col,
                                                                current,
                                                                offset_stack_pos,
                                                                parent_offset_index,
                                                                offset_depth);
  });
  return {current, offset_stack_pos};
}

/**
 * @brief Given a column, processed split buffers, and a metadata builder, populate
 * the metadata for this column in the builder, and return a tuple of:
 * column size, data offset, bitmask offset and null count.
 *
 * @param src column_view to create metadata from
 * @param current_info dst_buf_info pointer reference, pointing to this column's buffer info. 
 *                     This is a pointer reference because it is updated by this function as the 
 *                     columns's validity and data buffers are visited.
 * @param mb A metadata_builder instance to update with the columns's packed metadata
 * @param use_src_null_count True for the chunked_contiguous_split case where current_info
 *                           has invalid null count information. The null count should be taken
 *                           from `src` because this case is restricted to a single partition 
 *                           (no splits)
 * @returns a std::tuple<size_type, int64_t, int64_t, size_type> containing: 
 *          column size, data offset, bitmask offset, and null count.
 */
template <typename BufInfo>
std::tuple<size_type, int64_t, int64_t, size_type> build_output_column_metadata(
  column_view const& src,
  BufInfo& current_info,
  metadata_builder& mb,
  bool use_src_null_count)
{
  auto [bitmask_offset, null_count] = [&]() {
    // -1 means nullptr
    if (src.nullable()) {
      // TODO: so, offsets in the existing metadata are int64_t's
      // but here they are std::size_t
      int64_t const bitmask_offset =
        current_info->num_elements == 0
          ? (int64_t) -1 
          : (int64_t) current_info->dst_offset;

      // use_src_null_count is used for the chunked contig split case, where we have 
      // no splits: the null_count is just the source column's null_count
      size_type const null_count = use_src_null_count
                                     ? src.null_count()
                                     : (current_info->num_elements == 0
                                          ? 0
                                          : (current_info->num_rows - current_info->valid_count));
      ++current_info;
      return std::pair(bitmask_offset, null_count);
    }
    return std::pair((int64_t) -1, 0);
  }();
  
  // size/data pointer for the column
  auto col_size = current_info->num_elements;
  int64_t const data_offset =
    src.num_children() > 0 || col_size == 0 || src.head() == nullptr 
      ? (int64_t) -1 
      : (int64_t)current_info->dst_offset;

  mb.add_column_to_meta(src.type(),
                        (size_type)col_size,
                        (size_type)null_count,
                        data_offset,
                        bitmask_offset,
                        src.num_children());

  ++current_info;
  return std::make_tuple(col_size, data_offset, bitmask_offset, null_count);
}

/**
 * @brief Given a set of input columns and processed split buffers, produce
 * output columns.
 *
 * After performing the split we are left with 1 large buffer per incoming split
 * partition.  We need to traverse this buffer and distribute the individual
 * subpieces that represent individual columns and children to produce the final
 * output columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param info_begin Iterator of dst_buf_info structs containing information about each
 * copied buffer
 * @param out_begin Output iterator of column views
 * @param base_ptr Pointer to the base address of copied data for the working partition
 *
 * @returns new dst_buf_info iterator after processing this range of input columns
 */
template <typename InputIter, typename BufInfo, typename Output>
BufInfo build_output_columns(InputIter begin,
                             InputIter end,
                             BufInfo info_begin,
                             Output out_begin,
                             uint8_t const* const base_ptr,
                             metadata_builder& mb)
{
  auto current_info = info_begin;
  std::transform(begin, end, out_begin, [&current_info, base_ptr, &mb](column_view const& src) {
    size_type col_size, null_count;
    int64_t bitmask_offset;
    int64_t data_offset;
    std::tie(col_size, data_offset, bitmask_offset, null_count) =
      build_output_column_metadata<BufInfo>(src, current_info, mb, false);

    auto bitmask_ptr = base_ptr != nullptr && bitmask_offset != -1
                         ? reinterpret_cast<bitmask_type const*>(base_ptr + bitmask_offset)
                         : nullptr;

    // size/data pointer for the column
    uint8_t const* data_ptr =
      col_size == 0 || src.head() == nullptr ? nullptr : base_ptr + data_offset;

    // children
    auto children = std::vector<column_view>{};
    children.reserve(src.num_children());

    current_info = build_output_columns(
      src.child_begin(), 
      src.child_end(), 
      current_info, 
      std::back_inserter(children), 
      base_ptr, 
      mb);

    return column_view{
      src.type(), 
      col_size, 
      data_ptr, 
      bitmask_ptr, 
      null_count, 
      0, 
      std::move(children)};
  });

  return current_info;
}

/**
 * @brief Given a set of input columns, processed split buffers, and a metadata_builder,
 * append column metadata using the builder.
 *
 * After performing the split we are left with 1 large buffer per incoming split
 * partition.  We need to traverse this buffer and distribute the individual
 * subpieces that represent individual columns and children to produce the final
 * output columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param info_begin Iterator of dst_buf_info structs containing information about each
 * copied buffer
 * @param mb packed column metadata builder
 *
 * @returns new dst_buf_info iterator after processing this range of input columns
 */
template <typename InputIter, typename BufInfo>
BufInfo populate_metadata(InputIter begin,
                          InputIter end,
                          BufInfo info_begin,
                          metadata_builder& mb)
{
  auto current_info = info_begin;
  std::for_each(begin, end, [&current_info, &mb](column_view const& src) {
    build_output_column_metadata<BufInfo>(src, current_info, mb, true);

    // children
    current_info = populate_metadata(src.child_begin(), src.child_end(), current_info, mb);
  });

  return current_info;
}

/**
 * @brief Functor that retrieves the size of a destination buffer
 */
struct buf_size_functor {
  dst_buf_info const* ci;
  std::size_t operator() __device__(int index) { return ci[index].buf_size; }
};

/**
 * @brief Functor that retrieves the split "key" for a given output
 * buffer index.
 *
 * The key is simply the partition index.
 */
struct split_key_functor {
  int num_src_bufs;
  int operator() __device__(int buf_index)
  {
    return buf_index / num_src_bufs;
  }
};

/**
 * @brief Output iterator for writing values to the dst_offset field of the
 * dst_buf_info struct
 */
struct dst_offset_output_iterator {
  dst_buf_info* c;
  using value_type        = std::size_t;
  using difference_type   = std::size_t;
  using pointer           = std::size_t*;
  using reference         = std::size_t&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_offset_output_iterator operator+ __host__ __device__(int i) { return {c + i}; }

  void operator++ __host__ __device__() { c++; }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->dst_offset; }
};

/**
 * @brief Output iterator for writing values to the valid_count field of the
 * dst_buf_info struct
 */
struct dst_valid_count_output_iterator {
  dst_buf_info* c;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_valid_count_output_iterator operator+ __host__ __device__(int i)
  {
    return dst_valid_count_output_iterator{c + i};
  }

  void operator++ __host__ __device__() { c++; }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->valid_count; }
};

/**
 * @brief Functor for computing size of data elements for a given cudf type.
 *
 * Note: columns types which themselves inherently have no data (strings, lists,
 * structs) return 0.
 */
struct size_of_helper {
  template <typename T>
  constexpr std::enable_if_t<not is_fixed_width<T>(), int> __device__ operator()() const
  {
    return 0;
  }

  template <typename T>
  constexpr std::enable_if_t<is_fixed_width<T>(), int> __device__ operator()() const noexcept
  {
    return sizeof(cudf::device_storage_type_t<T>);
  }
};

/**
 * @brief Functor for returning the number of chunks an input buffer is being
 * subdivided into during the repartitioning step.
 *
 * Note: columns types which themselves inherently have no data (strings, lists,
 * structs) return 0.
 */
struct num_chunks_func {
  thrust::pair<std::size_t, std::size_t> const* chunks;
  __device__ std::size_t operator()(size_type i) const { return thrust::get<0>(chunks[i]); }
};

/**
 * @brief Get the size in bytes of a chunk described by `dst_buf_info`.
 */
struct chunk_byte_size_function {
  __device__ std::size_t operator()(const dst_buf_info& buf) const { 
    std::size_t const bytes =
      static_cast<std::size_t>(buf.num_elements) * static_cast<std::size_t>(buf.element_size);
    return util::round_up_unsafe(bytes, split_align);
  }
};

/**
 * @brief Get the input buffer index given the output buffer index.
 */
struct out_to_in_index_function {
  offset_type const* chunk_offsets;
  int num_bufs;
  __device__ int operator()(size_type i) const {
    return static_cast<size_type>(
            thrust::upper_bound(thrust::seq, chunk_offsets, chunk_offsets + num_bufs + 1, i) -
            chunk_offsets) - 1;
  }
};

};  // anonymous namespace

namespace detail {

// packed block of memory 1: split indices and src_buf_info structs
struct packed_split_indices_and_src_buf_info {
  explicit packed_split_indices_and_src_buf_info(cudf::table_view const& input,
                                                 std::vector<size_type> const& splits,
                                                 std::size_t num_partitions,
                                                 cudf::size_type num_src_bufs,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::mr::device_memory_resource* mr)
  {
    indices_size = cudf::util::round_up_safe((num_partitions + 1) * sizeof(size_type), split_align);
    src_buf_info_size = cudf::util::round_up_safe(num_src_bufs * sizeof(src_buf_info), split_align);

    // host-side
    h_indices_and_source_info = std::vector<uint8_t>(indices_size + src_buf_info_size);
    h_indices                 = reinterpret_cast<size_type*>(h_indices_and_source_info.data());
    h_src_buf_info =
      reinterpret_cast<src_buf_info*>(h_indices_and_source_info.data() + indices_size);

    // compute splits -> indices.
    // these are row numbers per split
    h_indices[0]              = 0;
    h_indices[num_partitions] = input.column(0).size();
    std::copy(splits.begin(), splits.end(), std::next(h_indices));

    // setup source buf info
    // TODO: ask: learn how this works
    setup_source_buf_info(input.begin(), input.end(), h_src_buf_info, h_src_buf_info);

    offset_stack_partition_size = compute_offset_stack_size(input.begin(), input.end());
    offset_stack_size           = offset_stack_partition_size * num_partitions * sizeof(size_type);
    // device-side
    // gpu-only : stack space needed for nested list offset calculation
    d_indices_and_source_info =
      rmm::device_buffer(indices_size + src_buf_info_size + offset_stack_size,
                         stream,
                         mr);
    d_indices      = reinterpret_cast<size_type*>(d_indices_and_source_info.data());
    d_src_buf_info = reinterpret_cast<src_buf_info*>(
      reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) + indices_size);
    d_offset_stack =
      reinterpret_cast<size_type*>(reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) +
                                   indices_size + src_buf_info_size);

    CUDF_CUDA_TRY(hipMemcpyAsync(
      d_indices, h_indices, indices_size + src_buf_info_size, hipMemcpyDefault, stream.value()));
  }

  size_type indices_size;
  std::size_t src_buf_info_size;
  std::size_t offset_stack_size;

  std::vector<uint8_t> h_indices_and_source_info;
  rmm::device_buffer d_indices_and_source_info;

  size_type* h_indices;
  src_buf_info* h_src_buf_info;

  int offset_stack_partition_size;
  size_type* d_indices;
  src_buf_info* d_src_buf_info;
  size_type* d_offset_stack;
};

// packed block of memory 2: partition buffer sizes and dst_buf_info structs
struct packed_partition_buf_size_and_dst_buf_info {
  packed_partition_buf_size_and_dst_buf_info(
    cudf::table_view const& input,
    std::vector<size_type> const& splits,
    std::size_t num_partitions,
    cudf::size_type num_src_bufs,
    std::size_t num_bufs,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr)
  {
    buf_sizes_size = cudf::util::round_up_safe(num_partitions * sizeof(std::size_t), split_align);
    dst_buf_info_size = cudf::util::round_up_safe(num_bufs * sizeof(dst_buf_info), split_align);
    // host-side
    h_buf_sizes_and_dst_info = std::vector<uint8_t>(buf_sizes_size + dst_buf_info_size);
    h_buf_sizes              = reinterpret_cast<std::size_t*>(h_buf_sizes_and_dst_info.data());
    h_dst_buf_info =
      reinterpret_cast<dst_buf_info*>(h_buf_sizes_and_dst_info.data() + buf_sizes_size);

    // device-side
    d_buf_sizes_and_dst_info = rmm::device_buffer(
      buf_sizes_size + dst_buf_info_size, stream, mr);
    d_buf_sizes = reinterpret_cast<std::size_t*>(d_buf_sizes_and_dst_info.data());

    //// destination buffer info
    d_dst_buf_info = reinterpret_cast<dst_buf_info*>(
      static_cast<uint8_t*>(d_buf_sizes_and_dst_info.data()) + buf_sizes_size);

    auto split_indices_and_src_buf_info = packed_split_indices_and_src_buf_info(
        input, splits, num_partitions, num_src_bufs, stream, mr);

    // this is a function because the lambda expression below
    initialize(num_src_bufs, num_bufs, split_indices_and_src_buf_info, stream, mr);
  }

  // buffer sizes and destination info (used in chunked copies)
  std::size_t buf_sizes_size;
  std::size_t dst_buf_info_size;

  std::vector<uint8_t> h_buf_sizes_and_dst_info;
  std::size_t* h_buf_sizes;
  dst_buf_info* h_dst_buf_info;

  rmm::device_buffer d_buf_sizes_and_dst_info;
  std::size_t* d_buf_sizes;
  dst_buf_info* d_dst_buf_info;

  void initialize(cudf::size_type num_src_bufs,
                  std::size_t num_bufs,
                  packed_split_indices_and_src_buf_info const& split_indices_and_src_buf_info,
                  rmm::cuda_stream_view stream,
                  rmm::mr::device_memory_resource* mr) {

    // TODO put split indices and this struct together?
    auto const d_src_buf_info        = split_indices_and_src_buf_info.d_src_buf_info;
    auto offset_stack_partition_size = split_indices_and_src_buf_info.offset_stack_partition_size;
    auto d_offset_stack              = split_indices_and_src_buf_info.d_offset_stack;
    auto d_indices                   = split_indices_and_src_buf_info.d_indices;

    // compute sizes of each column in each partition, including alignment.
    thrust::transform(
      rmm::exec_policy(stream, mr),
      thrust::make_counting_iterator<std::size_t>(0),
      thrust::make_counting_iterator<std::size_t>(num_bufs),
      d_dst_buf_info,
      [d_src_buf_info,
       offset_stack_partition_size,
       d_offset_stack,
       d_indices,
       num_src_bufs] __device__(std::size_t t) {
        int const split_index   = t / num_src_bufs;
        int const src_buf_index = t % num_src_bufs;
        auto const& src_info    = d_src_buf_info[src_buf_index];

        // apply nested offsets (lists and string columns).
        //
        // We can't just use the incoming row indices to figure out where to read from in a
        // nested list situation.  We have to apply offsets every time we cross a boundary
        // (list or string).  This loop applies those offsets so that our incoming row_index_start
        // and row_index_end get transformed to our final values.
        //
        int const stack_pos =
          src_info.offset_stack_pos + (split_index * offset_stack_partition_size);
        size_type* offset_stack  = &(d_offset_stack[stack_pos]);
        int parent_offsets_index = src_info.parent_offsets_index;
        int stack_size           = 0;
        int root_column_offset   = src_info.column_offset;

        // TODO: ask: what is this loop doing
        while (parent_offsets_index >= 0) {
          offset_stack[stack_size++] = parent_offsets_index;
          root_column_offset         = d_src_buf_info[parent_offsets_index].column_offset;
          parent_offsets_index       = d_src_buf_info[parent_offsets_index].parent_offsets_index;
        }
        // make sure to include the -column- offset on the root column in our calculation.
        int row_start = d_indices[split_index] + root_column_offset;
        int row_end   = d_indices[split_index + 1] + root_column_offset;
        while (stack_size > 0) {
          stack_size--;
          auto const offsets = d_src_buf_info[offset_stack[stack_size]].offsets;
          // this case can happen when you have empty string or list columns constructed with
          // empty_like()
          if (offsets != nullptr) {
            row_start = offsets[row_start];
            row_end   = offsets[row_end];
          }
        }

        // final element indices and row count
        int const out_element_index = src_info.is_validity ? row_start / 32 : row_start;
        int const num_rows          = row_end - row_start;
        // if I am an offsets column, all my values need to be shifted
        int const value_shift = src_info.offsets == nullptr ? 0 : src_info.offsets[row_start];
        // if I am a validity column, we may need to shift bits
        int const bit_shift = src_info.is_validity ? row_start % 32 : 0;
        // # of rows isn't necessarily the same as # of elements to be copied.
        auto const num_elements = [&]() {
          if (src_info.offsets != nullptr && num_rows > 0) {
            return num_rows + 1;
          } else if (src_info.is_validity) {
            return (num_rows + 31) / 32;
          }
          return num_rows;
        }();
        int const element_size = cudf::type_dispatcher(data_type{src_info.type}, size_of_helper{});
        std::size_t const bytes =
          static_cast<std::size_t>(num_elements) * static_cast<std::size_t>(element_size);

        return dst_buf_info{util::round_up_unsafe(bytes, split_align),
                            num_elements,
                            element_size,
                            num_rows,
                            out_element_index,
                            0,
                            value_shift,
                            bit_shift,
                            src_info.is_validity ? 1 : 0,
                            src_buf_index,
                            split_index};
      });

    // compute total size of each partition
    // key is split index
    {
      auto keys = cudf::detail::make_counting_transform_iterator(
        0, split_key_functor{static_cast<int>(num_src_bufs)});
      auto values =
        cudf::detail::make_counting_transform_iterator(0, buf_size_functor{d_dst_buf_info});

      thrust::reduce_by_key(rmm::exec_policy(stream, mr),
                            keys,
                            keys + num_bufs,
                            values,
                            thrust::make_discard_iterator(),
                            d_buf_sizes);
    }

    // compute start offset for each output buffer for each split
    {
      auto keys = cudf::detail::make_counting_transform_iterator(
        0, split_key_functor{static_cast<int>(num_src_bufs)});
      auto values =
        cudf::detail::make_counting_transform_iterator(0, buf_size_functor{d_dst_buf_info});

      thrust::exclusive_scan_by_key(rmm::exec_policy(stream, mr),
                                    keys,
                                    keys + num_bufs,
                                    values,
                                    dst_offset_output_iterator{d_dst_buf_info},
                                    std::size_t{0});
    }
  
    // DtoH buf sizes and col info back to the host
    CUDF_CUDA_TRY(hipMemcpyAsync(h_buf_sizes,
                                  d_buf_sizes,
                                  buf_sizes_size + dst_buf_info_size,
                                  hipMemcpyDefault,
                                  stream.value()));

    stream.synchronize();
  }
};

// Packed block of memory 3:
// Pointers to source and destination buffers (and stack space on the
// gpu for offset computation)
struct packed_src_and_dst_pointers {
  packed_src_and_dst_pointers(cudf::table_view const& input,
                              std::size_t num_partitions,
                              cudf::size_type num_src_bufs,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr):
                              stream(stream)
  {
    src_bufs_size =
      cudf::util::round_up_safe(num_src_bufs * sizeof(uint8_t*), split_align);
    dst_bufs_size =
      cudf::util::round_up_safe(num_partitions * sizeof(uint8_t*), split_align);

    // host-side
    h_src_and_dst_buffers = std::vector<uint8_t>(src_bufs_size + dst_bufs_size);
    h_src_bufs = reinterpret_cast<uint8_t const**>(h_src_and_dst_buffers.data());
    h_dst_bufs = reinterpret_cast<uint8_t**>(h_src_and_dst_buffers.data() + src_bufs_size);

    // device-side
    d_src_and_dst_buffers = rmm::device_buffer(
      src_bufs_size + dst_bufs_size, stream, mr);
    d_src_bufs = reinterpret_cast<uint8_t const**>(d_src_and_dst_buffers.data());
    d_dst_bufs = reinterpret_cast<uint8_t**>(
      reinterpret_cast<uint8_t*>(d_src_and_dst_buffers.data()) + src_bufs_size);

    // setup src buffers
    setup_src_buf_data(input.begin(), input.end(), h_src_bufs);
  }

  //
  // We execute this for every chunk rather than once in the regular contiguous_split case.
  // This is mildly wasteful since the "src" info will already be in place - only the destination
  // pointer is going to change. That said, the data is small.
  //
  void copy_to_device() {
    // TODO: make this copy the root device buffer/host buffer.. slightly cleaner
    CUDF_CUDA_TRY(hipMemcpyAsync(
      d_src_bufs, h_src_bufs, src_bufs_size + dst_bufs_size, hipMemcpyDefault, stream.value()));
  }

  const rmm::cuda_stream_view stream;

  std::vector<uint8_t> h_src_and_dst_buffers;
  rmm::device_buffer d_src_and_dst_buffers;
  std::size_t src_bufs_size; 
  std::size_t dst_bufs_size; 
  const uint8_t** h_src_bufs;
  const uint8_t** d_src_bufs;
  uint8_t** h_dst_bufs;
  uint8_t** d_dst_bufs;
};

struct iteration_state {
  iteration_state(rmm::device_uvector<dst_buf_info> _d_chunked_dst_buf_info,
                  std::vector<std::size_t> _h_num_buffs_per_key,
                  std::vector<std::size_t> _h_size_of_buffs_per_key,
                  std::size_t total_size,
                  int num_expected_copies)
    : num_iterations(num_expected_copies),
      current_iteration(0),
      starting_buff(0),
      d_chunked_dst_buf_info(std::move(_d_chunked_dst_buf_info)),
      h_num_buffs_per_key(std::move(_h_num_buffs_per_key)),
      h_size_of_buffs_per_key(std::move(_h_size_of_buffs_per_key)),
      total_size(total_size)
  {
  }
  
  std::pair<std::size_t, std::size_t> get_current_starting_index_and_buff_count() const {
    CUDF_EXPECTS(current_iteration < num_iterations, 
      "current_iteration cannot exceed than num_iterations");
    auto count_for_current = h_num_buffs_per_key[current_iteration];
    return std::make_pair(starting_buff, count_for_current);
  }
  
  std::size_t advance_iteration() { 
    CUDF_EXPECTS(current_iteration < num_iterations, 
      "current_iteration cannot exceed than num_iterations");
    std::size_t bytes_copied = h_size_of_buffs_per_key[current_iteration];
    starting_buff += h_num_buffs_per_key[current_iteration];
    ++current_iteration;
    return bytes_copied;
  }

  bool has_more_copies() const { 
    return current_iteration < num_iterations;
  }

  rmm::device_uvector<dst_buf_info> d_chunked_dst_buf_info;
  std::size_t total_size;

private:
  int num_iterations;
  int current_iteration;
  std::size_t starting_buff;
  std::vector<std::size_t> h_num_buffs_per_key;
  std::vector<std::size_t> h_size_of_buffs_per_key;
};

// TODO: this function is badly named, it doesn't return a dst_buf_info
std::unique_ptr<iteration_state> get_dst_buf_info(
  rmm::device_uvector<thrust::pair<std::size_t, std::size_t>>& chunks,
  rmm::device_uvector<offset_type>& chunk_offsets,
  int num_chunks,
  int num_bufs,
  int num_src_bufs,
  dst_buf_info* d_orig_dst_buf_info,
  std::size_t const* const  h_buf_sizes,
  std::size_t user_buffer_size,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr) {

  auto out_to_in_index = out_to_in_index_function{chunk_offsets.begin(), num_bufs};

  auto iter = thrust::make_counting_iterator(0);

  // load up the chunks as d_dst_buf_info
  rmm::device_uvector<dst_buf_info> d_chunked_dst_buf_info(num_chunks, stream, mr);

  thrust::for_each(
    rmm::exec_policy(stream, mr),
    iter,
    iter + num_chunks,
    [d_orig_dst_buf_info,
     d_chunked_dst_buf_info = d_chunked_dst_buf_info.begin(),
     chunks         = chunks.begin(),
     chunk_offsets  = chunk_offsets.begin(),
     num_bufs,
     num_src_bufs,
     out_to_in_index] __device__(size_type i) {
      size_type const in_buf_index = out_to_in_index(i);
      size_type const chunk_index  = i - chunk_offsets[in_buf_index];
      auto const chunk_size        = thrust::get<1>(chunks[in_buf_index]);
      dst_buf_info const& in       = d_orig_dst_buf_info[in_buf_index];

      // adjust info
      dst_buf_info& out = d_chunked_dst_buf_info[i];
      out.element_size  = in.element_size;
      out.value_shift   = in.value_shift;
      out.bit_shift     = in.bit_shift;
      out.valid_count =
        in.valid_count;  // valid count will be set to 1 if this is a validity buffer
      out.src_buf_index = in.src_buf_index;
      out.dst_buf_index = in.dst_buf_index;

      size_type const elements_per_chunk =
        out.element_size == 0 ? 0 : chunk_size / out.element_size;
      out.num_elements = ((chunk_index + 1) * elements_per_chunk) > in.num_elements
                           ? in.num_elements - (chunk_index * elements_per_chunk)
                           : elements_per_chunk;

      size_type const rows_per_chunk =
        // if this is a validity buffer, each element is a bitmask_type, which
        // corresponds to 32 rows.
        out.valid_count > 0
          ? elements_per_chunk * static_cast<size_type>(cudf::detail::size_in_bits<bitmask_type>())
          : elements_per_chunk;
      out.num_rows = ((chunk_index + 1) * rows_per_chunk) > in.num_rows
                       ? in.num_rows - (chunk_index * rows_per_chunk)
                       : rows_per_chunk;

      out.src_element_index = in.src_element_index + (chunk_index * elements_per_chunk);
      out.dst_offset        = in.dst_offset + (chunk_index * chunk_size);

      // out.bytes and out.buf_size are unneeded here because they are only used to
      // calculate real output buffer sizes. the data we are generating here is
      // purely intermediate for the purposes of doing more uniform copying of data
      // underneath the final structure of the output
    });

  if (user_buffer_size != 0) {
    // copy the chunk sizes back to host
    std::vector<std::size_t> h_sizes(num_chunks);
    {
      rmm::device_uvector<std::size_t> sizes(num_chunks, stream, mr);
      thrust::transform(rmm::exec_policy(stream, mr),
                        d_chunked_dst_buf_info.begin(),
                        d_chunked_dst_buf_info.end(),
                        sizes.begin(),
                        chunk_byte_size_function());
      
      CUDF_CUDA_TRY(hipMemcpyAsync(h_sizes.data(),
                                    sizes.data(),
                                    sizeof(std::size_t) * sizes.size(),
                                    hipMemcpyDefault,
                                    stream.value()));

      // the next part is working on the CPU, so we want to synchronize here
      stream.synchronize();
    }

    // TODO: can this be simplified + the device memory allocation  done after it
    // compute the chunks size and offsets
    std::vector<std::size_t> offset_per_chunk(num_chunks);
    std::vector<std::size_t> num_chunks_per_split;
    std::vector<std::size_t> size_of_chunks_per_split;
    std::vector<std::size_t> accum_size_per_split;
    std::size_t accum_size = 0;
    {
      std::size_t current_split_num_chunks = 0;
      std::size_t current_split_size       = 0;

      int current_split = 0;
      for (std::size_t i = 0; i < h_sizes.size(); ++i) {
        auto curr_size = h_sizes[i];
        if (current_split_size + curr_size > user_buffer_size) {
          num_chunks_per_split.push_back(current_split_num_chunks);
          size_of_chunks_per_split.push_back(current_split_size);
          accum_size_per_split.push_back(accum_size);
          current_split_num_chunks = 0;
          current_split_size       = 0;
          ++current_split;
        }
        offset_per_chunk[i] = current_split;
        current_split_size += curr_size;
        accum_size += curr_size;
        ++current_split_num_chunks;
      }
      if (current_split_num_chunks > 0) {
        num_chunks_per_split.push_back(current_split_num_chunks);
        size_of_chunks_per_split.push_back(current_split_size);
        accum_size_per_split.push_back(accum_size);
      }
    }

    // apply changed offset
    {
      rmm::device_uvector<std::size_t> d_offset_per_chunk(num_chunks, stream, mr);
      rmm::device_uvector<std::size_t> d_accum_size_per_split(accum_size_per_split.size(), stream, mr);

      CUDF_CUDA_TRY(hipMemcpyAsync(
        d_offset_per_chunk.data(), offset_per_chunk.data(), num_chunks * sizeof(std::size_t), hipMemcpyDefault, stream.value()));
      CUDF_CUDA_TRY(hipMemcpyAsync(
        d_accum_size_per_split.data(), accum_size_per_split.data(), accum_size_per_split.size() * sizeof(std::size_t), hipMemcpyDefault, stream.value()));

      // we want to update the offset of chunks in the second to last copy
      auto num_chunks_in_first_split = num_chunks_per_split[0];
      auto iter = thrust::make_counting_iterator(num_chunks_in_first_split);
      thrust::for_each(rmm::exec_policy(stream, mr),
                       iter,
                       iter + num_chunks - num_chunks_in_first_split,
                       [d_chunked_dst_buf_info = d_chunked_dst_buf_info.begin(),
                        d_accum_size_per_split = d_accum_size_per_split.begin(),
                        d_offset_per_chunk = d_offset_per_chunk.begin()] __device__(size_type i) {
                          auto split = d_offset_per_chunk[i];
                          d_chunked_dst_buf_info[i].dst_offset -= d_accum_size_per_split[split - 1];
                       });
    }
    return std::make_unique<iteration_state>(
      std::move(d_chunked_dst_buf_info), 
      std::move(num_chunks_per_split),
      std::move(size_of_chunks_per_split),
      accum_size,
      num_chunks_per_split.size());

  } else {
    std::vector<std::size_t> regular = { (std::size_t)num_chunks } ;
    std::vector<std::size_t> regular_sizes = { h_buf_sizes[0]} ;
    return std::make_unique<iteration_state>(
      std::move(d_chunked_dst_buf_info), 
      std::move(regular), 
      std::move(regular_sizes),
      h_buf_sizes[0],
      1);
  }
}

void copy_data(int num_chunks_to_copy,
               int starting_chunk,
               uint8_t const** d_src_bufs,
               uint8_t** d_dst_bufs,
               rmm::device_uvector<dst_buf_info>& d_dst_buf_info,
               rmm::cuda_stream_view stream)
{
  constexpr size_type block_size = 256;
  copy_partitions<block_size><<<num_chunks_to_copy, block_size, 0, stream.value()>>>(
    d_src_bufs, d_dst_bufs, d_dst_buf_info.data() + starting_chunk);
}

std::size_t get_num_partitions(std::vector<size_type> const& splits) {
  return splits.size() + 1;
}

struct chunk_infos {
  chunk_infos(rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> _chunks,
              rmm::device_uvector<offset_type> _chunk_offsets)
    : chunks(std::move(_chunks)), chunk_offsets(std::move(_chunk_offsets))
  {
  }

  rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> chunks;
  rmm::device_uvector<offset_type> chunk_offsets;
};

bool check_inputs(cudf::table_view const& input, std::vector<size_type> const& splits) 
{
  if (input.num_columns() == 0) {
    // TODO: does this work for no columns?
    return true;
  }
  if (splits.size() > 0) {
    CUDF_EXPECTS(splits.back() <= input.column(0).size(),
                 "splits can't exceed size of input columns");
  }
  {
    size_type begin = 0;
    for (std::size_t i = 0; i < splits.size(); i++) {
      size_type end = splits[i];
      CUDF_EXPECTS(begin >= 0, "Starting index cannot be negative.");
      CUDF_EXPECTS(end >= begin, "End index cannot be smaller than the starting index.");
      CUDF_EXPECTS(end <= input.column(0).size(), "Slice range out of bounds.");
      begin = end;
    }
  }
  // if inputs are empty, just return num_partitions empty tables
  return input.column(0).size() == 0;
}

struct contiguous_split_state {

  static const std::size_t desired_chunk_size = 1 * 1024 * 1024;

  contiguous_split_state(cudf::table_view const& input,
                         std::size_t user_buffer_size,
                         rmm::cuda_stream_view stream,
                         rmm::mr::device_memory_resource* mr)
    : contiguous_split_state(input, {}, user_buffer_size, stream, mr)
  {
  }

  contiguous_split_state(cudf::table_view const& input,
                         std::vector<size_type> const& splits,
                         rmm::cuda_stream_view stream,
                         rmm::mr::device_memory_resource* mr)
    : contiguous_split_state(input, splits, 0, stream, mr)
  {
  }

  contiguous_split_state(cudf::table_view const& input,
                         std::vector<size_type> const& splits,
                         std::size_t user_buffer_size,
                         rmm::cuda_stream_view stream,
                         rmm::mr::device_memory_resource* mr)
    : input(input),
      user_buffer_size(user_buffer_size),
      stream(stream),
      mr(mr)
  {
    is_empty       = check_inputs(input, splits);
    num_partitions = get_num_partitions(splits);
    num_src_bufs   = count_src_bufs(input.begin(), input.end());
    num_bufs       = num_src_bufs * num_partitions;

    if (is_empty) { return; }

    partition_buf_size_and_dst_buf_info =
      std::make_unique<detail::packed_partition_buf_size_and_dst_buf_info>(
        input,
        splits,
        num_partitions,
        num_src_bufs,
        num_bufs,
        stream,
        mr);

    src_and_dst_pointers = std::make_unique<packed_src_and_dst_pointers>(
      input, num_partitions, num_src_bufs, stream, mr);

    // allocate output partition buffers, if needed
    if (user_buffer_size == 0) {
      out_buffers.reserve(num_partitions);
      std::transform(partition_buf_size_and_dst_buf_info->h_buf_sizes,
                     partition_buf_size_and_dst_buf_info->h_buf_sizes + num_partitions,
                     std::back_inserter(out_buffers),
                     [stream = stream, mr = mr](std::size_t bytes) {
                       return rmm::device_buffer{bytes, stream, mr};
                     });
      std::transform(
        out_buffers.begin(), out_buffers.end(), src_and_dst_pointers->h_dst_bufs, [](auto& buf) {
          return static_cast<uint8_t*>(buf.data());
        });
    } 

    compute_chunks();
  }

  bool has_next() const { 
    return !is_empty && internal_iter_state->has_more_copies(); 
  }

  // TODO: this only works for chunked
  std::size_t get_total_contiguous_size() const {
    return is_empty ? 0 : internal_iter_state->total_size;
  }

  void compute_chunks()
  {
    //
    // CHUNKED F:  So this is where the "real" array of destination buffers to be copied
    // (_d_dst_buf_info)
    //             is further partitioned into smaller chunks.  these more granular chunks are what
    //             gets passed to the copy kernel.  So ultimately what needs to happen is
    //               - refactor out this whole block so it can be called by the chunked packer
    //               seperately.
    //               - verify that the ordering of the chunks is linear with the overall output
    //                 buffer. that is,
    // TODO: I do not think I have succeeded at verifying this:
    //                 d_dst_buf_info[N] is copying to the destination location exactly where
    //                 d_dst_buf_info[N-1] ends.
    //               - the intermediate chunked struct would store d_dst_buf_info across calls as
    //                 well as the count of buffers we've processed so far.
    //               - when it's time to pack another chunk, we are given a buffer and a size from
    //                 the caller.  we search
    //                 forward from the last used pos in the d_dst_buf_info array until we cross the
    //                 output size (there's a good example of doing this quickly in
    //                 cpp/io/src/parquet/reader_impl_preprocess.cu -> find_splits. we call
    //                 copy_partitions on that subset of dst_buf_infos.
    //               - I -think- the only thing that will need to get updated for each chunk is
    //                 dst_buf_info::dst_offset.
    //                 Everything else that would need to be changed should be 0 (value_shift,
    //                 bit_shift, etc) because we're only outputting 1 "real" partition at the end
    //                 of the day
    //               - In the packed case, after computing d_dst_buf_info, do a scan on all the
    //                 sizes to generate
    //                 cumulative sizes so we can determine what chunks to read. This will get
    //                 stored in the intermediate data.
    //
    // Since we parallelize at one block per copy, we are vulnerable to situations where we
    // have small numbers of copies to do (a combination of small numbers of splits and/or columns),
    // so we will take the actual set of outgoing source/destination buffers and further partition
    // them into much smaller chunks in order to drive up the number of blocks and overall
    // occupancy.
    
    // TODO: should probably call this something differently instead of just chunks.
    rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> chunks(num_bufs, stream, mr);
    auto& d_dst_buf_info = partition_buf_size_and_dst_buf_info->d_dst_buf_info;
    auto desired_chunk_size = contiguous_split_state::desired_chunk_size;
    thrust::transform(
      rmm::exec_policy(stream, mr),
      d_dst_buf_info,
      d_dst_buf_info + num_bufs,
      chunks.begin(),
      [desired_chunk_size] __device__(
        dst_buf_info const& buf) -> thrust::pair<std::size_t, std::size_t> {
        // Total bytes for this incoming partition
        std::size_t const bytes =
          static_cast<std::size_t>(buf.num_elements) * static_cast<std::size_t>(buf.element_size);

        // This clause handles nested data types (e.g. list or string) that store no data in the row
        // columns, only in their children.
        if (bytes == 0) { return {1, 0}; }

        // The number of chunks we want to subdivide this buffer into
        std::size_t const num_chunks = std::max(
          std::size_t{1}, util::round_up_unsafe(bytes, desired_chunk_size) / desired_chunk_size);

        // NOTE: leaving chunk size as a separate parameter for future tuning
        // possibilities, even though in the current implementation it will be a
        // constant.
        return {num_chunks, desired_chunk_size};
      });


    std::size_t& my_num_bufs = num_bufs;
    rmm::device_uvector<offset_type> chunk_offsets(num_bufs + 1, stream, mr);
    auto buf_count_iter = cudf::detail::make_counting_transform_iterator(
      0, [my_num_bufs, num_chunks = num_chunks_func{chunks.begin()}] __device__(size_type i) {
        return i == my_num_bufs ? 0 : num_chunks(i);
      });

    thrust::exclusive_scan(rmm::exec_policy(stream, mr),
                           buf_count_iter,
                           buf_count_iter + num_bufs + 1,
                           chunk_offsets.begin(),
                           0);

    // used during the copy
    computed_chunks = std::make_unique<chunk_infos>(std::move(chunks), std::move(chunk_offsets));

    // HtoD src and dest buffers
    src_and_dst_pointers->copy_to_device();

    auto const num_chunks_iter = cudf::detail::make_counting_transform_iterator(
      0, num_chunks_func{computed_chunks->chunks.begin()});
    size_type const new_buf_count = thrust::reduce(
      rmm::exec_policy(stream, mr), 
      num_chunks_iter, 
      num_chunks_iter + computed_chunks->chunks.size());

    internal_iter_state = get_dst_buf_info(computed_chunks->chunks,
                                           computed_chunks->chunk_offsets,
                                           new_buf_count,
                                           num_bufs,
                                           num_src_bufs,
                                           partition_buf_size_and_dst_buf_info->d_dst_buf_info,
                                           partition_buf_size_and_dst_buf_info->h_buf_sizes,
                                           user_buffer_size,
                                           stream,
                                           mr);
  }

  std::vector<packed_table> contiguous_split()
  {
    CUDF_FUNC_RANGE()
    CUDF_EXPECTS(user_buffer_size == 0, "Cannot contiguous split with a user buffer");
    if (is_empty || input.num_columns() == 0) { return make_packed_tables(); }

    // apply the chunking.
    auto const num_chunks = cudf::detail::make_counting_transform_iterator(
      0, num_chunks_func{computed_chunks->chunks.begin()});
    size_type const new_buf_count = thrust::reduce(
      rmm::exec_policy(stream, mr), num_chunks, num_chunks + computed_chunks->chunks.size());

    // these "orig" dst_buf_info pointers describe the prior-to-chunking destination
    // buffers per partition
    auto d_orig_dst_buf_info = partition_buf_size_and_dst_buf_info->d_dst_buf_info;
    auto h_orig_dst_buf_info = partition_buf_size_and_dst_buf_info->h_dst_buf_info;

    // perform the copy.
    copy_data(new_buf_count,
              0 /* starting at buffer 0*/,
              src_and_dst_pointers->d_src_bufs,
              src_and_dst_pointers->d_dst_bufs,
              internal_iter_state->d_chunked_dst_buf_info,
              stream);

    // postprocess valid_counts: apply the valid counts computed by copy_data for each 
    // chunk back to the original dst_buf_infos
    auto keys = cudf::detail::make_counting_transform_iterator(
      0, out_to_in_index_function{computed_chunks->chunk_offsets.begin(), (int)num_bufs});

    auto values = thrust::make_transform_iterator(internal_iter_state->d_chunked_dst_buf_info.begin(),
      [] __device__(dst_buf_info const& info) { return info.valid_count; });

    thrust::reduce_by_key(rmm::exec_policy(stream, mr),
                          keys,
                          keys + new_buf_count,
                          values,
                          thrust::make_discard_iterator(),
                          dst_valid_count_output_iterator{d_orig_dst_buf_info});

    CUDF_CUDA_TRY(hipMemcpyAsync(h_orig_dst_buf_info,
                                  d_orig_dst_buf_info,
                                  partition_buf_size_and_dst_buf_info->dst_buf_info_size,
                                  hipMemcpyDefault,
                                  stream.value()));

    stream.synchronize();

    // not necessary for the non-chunked case, but it makes it so further calls to has_next
    // return false, just in case
    internal_iter_state->advance_iteration();

    return make_packed_tables();
  }

  cudf::size_type contigous_split_chunk(cudf::device_span<uint8_t> const& user_buffer)
  {
    CUDF_FUNC_RANGE()
    CUDF_EXPECTS(user_buffer.size() == user_buffer_size, 
      "Cannot use a device span smaller than the output buffer size configured at instantiation!");
    CUDF_EXPECTS(has_next(),
      "Cannot call contiguos_split_chunk with has_next() == false!");
    // prep the target location
    src_and_dst_pointers->h_dst_bufs[0] = user_buffer.data();
    src_and_dst_pointers->copy_to_device();

    std::size_t starting_buff, num_chunks_to_copy;
    std::tie(starting_buff, num_chunks_to_copy) =
      internal_iter_state->get_current_starting_index_and_buff_count();

    // perform the copy.
    copy_data(num_chunks_to_copy,
              starting_buff,
              src_and_dst_pointers->d_src_bufs,
              src_and_dst_pointers->d_dst_bufs,
              internal_iter_state->d_chunked_dst_buf_info,
              stream);

    // We do not need to post-process null counts since the null count info is 
    // taken from the source table in the contigous_split_chunk case (no splits)
    return internal_iter_state->advance_iteration();
  }

  std::vector<packed_table> make_empty_packed_table() {
    // sanitize the inputs (to handle corner cases like sliced tables)
    std::vector<std::unique_ptr<column>> empty_columns;
    empty_columns.reserve(input.num_columns());
    std::transform(
      input.begin(), input.end(), std::back_inserter(empty_columns), [](column_view const& col) {
        return cudf::empty_like(col);
      });
    std::vector<cudf::column_view> empty_column_views;
    empty_column_views.reserve(input.num_columns());
    std::transform(empty_columns.begin(),
                   empty_columns.end(),
                   std::back_inserter(empty_column_views),
                   [](std::unique_ptr<column> const& col) { return col->view(); });
    table_view empty_inputs(empty_column_views);

    // build the empty results
    std::vector<packed_table> result;
    result.reserve(num_partitions);
    auto iter = thrust::make_counting_iterator(0);
    std::transform(iter,
                   iter + num_partitions,
                   std::back_inserter(result),
                   [&empty_inputs](int partition_index) {
                     return packed_table{
                       empty_inputs,
                       packed_columns{std::make_unique<packed_columns::metadata>(pack_metadata(
                                        empty_inputs, static_cast<uint8_t const*>(nullptr), 0)),
                                      std::make_unique<rmm::device_buffer>()}};
                   });

    return result;
  }

  std::unique_ptr<packed_columns::metadata> make_packed_column_metadata()
  {
    CUDF_EXPECTS(num_partitions == 1, 
      "make_packed_column_metadata supported only without splits");

    if (input.num_columns() == 0) { return std::unique_ptr<packed_columns::metadata>(); }

    if (is_empty) { 
      // this is a bit ugly, but it was done to re-use make_empty_packed_table between the
      // regular contiguous_split and chunked_contiguous_split cases.
      auto empty_packed_tables = std::move(make_empty_packed_table()[0]);
      return std::move(empty_packed_tables.data.metadata_);
    }

    auto& h_dst_buf_info  = partition_buf_size_and_dst_buf_info->h_dst_buf_info;
    auto cur_dst_buf_info = h_dst_buf_info;
    metadata_builder mb(input.num_columns());

    // traverse the buffers and build the columns.
    populate_metadata(input.begin(), input.end(), cur_dst_buf_info, mb);

    return std::make_unique<packed_columns::metadata>(std::move(mb.build()));
  }

  std::vector<packed_table> make_packed_tables()
  {
    if (input.num_columns() == 0) { return std::vector<packed_table>(); }
    if (is_empty){ return make_empty_packed_table(); }
    std::vector<packed_table> result;
    result.reserve(num_partitions);
    std::vector<column_view> cols;
    cols.reserve(input.num_columns());

    auto& h_dst_buf_info = partition_buf_size_and_dst_buf_info->h_dst_buf_info;
    auto& h_dst_bufs     = src_and_dst_pointers->h_dst_bufs;

    auto cur_dst_buf_info = h_dst_buf_info;
    for (std::size_t idx = 0; idx < num_partitions; idx++) {
      // traverse the buffers and build the columns.
      metadata_builder mb(input.num_columns());
      cur_dst_buf_info = cudf::build_output_columns(
        input.begin(), 
        input.end(), 
        cur_dst_buf_info, 
        std::back_inserter(cols), 
        h_dst_bufs[idx], 
        mb);

      // pack the columns
      cudf::table_view t{cols};
      result.push_back(packed_table{
        t,
        packed_columns{
          std::make_unique<packed_columns::metadata>(mb.build()),
          std::make_unique<rmm::device_buffer>(std::move(out_buffers[idx]))}});

      cols.clear();
    }

    return result;
  }

  cudf::table_view const& input;
  rmm::cuda_stream_view stream;
  rmm::mr::device_memory_resource* mr;

  std::size_t num_partitions;

  // number of source buffers including children * number of splits
  std::size_t num_bufs;

  // number of source buffers including children
  size_type num_src_bufs;

  std::unique_ptr<packed_partition_buf_size_and_dst_buf_info> partition_buf_size_and_dst_buf_info;

  std::unique_ptr<packed_src_and_dst_pointers> src_and_dst_pointers;

  //
  // State around the iterator pattern
  //

  // the chunk data computed once on initialization
  std::unique_ptr<chunk_infos> computed_chunks;

  // whether the table was empty to begin with
  // TODO: ask: empty columns vs columns but no rows
  bool is_empty;

  std::unique_ptr<iteration_state> internal_iter_state;

  // two result buffer types are allowed:
  //  - user provided: as the name implies, the user has provided a buffer that must be at least
  //  1MB.
  //    contiguous_split will behave in a "chunked" mode in this scenario, as it will contiguously
  //    copy up until the user's buffer size limit, exposing a next() call for the user to invoke.
  //    Note that in this mode, contig split is not partitioning the original table, it is instead
  //    only placing cuDF buffers contigously in the user's bounce buffer.
  //
  //  - out_buffers: when the user doesn't provide their own buffer, contiguous_split will allocate
  //    a buffer per partition and will place contiguous results in each element of out_buffers.
  //
  std::vector<rmm::device_buffer> out_buffers;

  std::size_t user_buffer_size;
};

std::vector<packed_table> contiguous_split(cudf::table_view const& input,
                                           std::vector<size_type> const& splits,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
{
  auto state = contiguous_split_state(input, splits, stream, mr);
  return state.contiguous_split();
}

};  // namespace detail

std::vector<packed_table> contiguous_split(cudf::table_view const& input,
                                           std::vector<size_type> const& splits,
                                           rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contiguous_split(input, splits, cudf::get_default_stream(), mr);
}

chunked_contiguous_split::chunked_contiguous_split(cudf::table_view const& input,
                                                   std::size_t user_buffer_size,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  state = std::make_unique<detail::contiguous_split_state>(input, user_buffer_size, stream, mr);
}

// required for the unique_ptr to work with a non-complete type (contiguous_split_state)
chunked_contiguous_split::~chunked_contiguous_split() = default;
    
std::size_t chunked_contiguous_split::get_total_contiguous_size() const {
  return state->get_total_contiguous_size();
}

bool chunked_contiguous_split::has_next() const { return state->has_next(); }

std::size_t chunked_contiguous_split::next(cudf::device_span<uint8_t> const& user_buffer)
{
  return state->contigous_split_chunk(user_buffer);
}

std::unique_ptr<packed_columns::metadata> chunked_contiguous_split::make_packed_columns() const
{
  return state->make_packed_column_metadata();
}

std::unique_ptr<chunked_contiguous_split> make_chunked_contiguous_split(
  cudf::table_view const& input,
  std::size_t user_buffer_size,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(user_buffer_size >= detail::contiguous_split_state::desired_chunk_size,
    "The output buffer size must be at least 1MB in size");
  return std::make_unique<chunked_contiguous_split>(
    input, user_buffer_size, cudf::get_default_stream(), mr);
}

};  // namespace cudf
